
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>     // srand, rand
#include <ctime>       // time
#include <sys/time.h>  // get_wall_time

#define IMDEP 256
#define SIZE (100*1024*1024) // 100 MB

#define NBLOCKS 256
#define THREADS_PER_BLOCK 256

const int numRuns = 10;

double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        printf("Error en la medicion de tiempo CPU!!\n");
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

void* inicializarImagen(unsigned long nBytes){
        unsigned char * img = (unsigned char*) malloc( nBytes );
        for(unsigned long i = 0; i<nBytes; i++){
                img[i] = rand() % IMDEP;
        }
        return img;
}

void histogramaCPU(unsigned char* img, unsigned long nBytes, unsigned int* histo){
        for(int i = 0; i<IMDEP; i++)
                histo[i] = 0;//Inicializacion
        for(unsigned long i = 0; i<nBytes; i++){
                histo[ img[i] ]++;
        }
        printf("Tiempo de CPU (s): %.4lf\n", 0.0);
}

long calcularCheckSum(unsigned int* histo){
        long checkSum = 0;
        for(int i = 0; i<IMDEP; i++){
                checkSum += histo[i];
        }
        return checkSum;
}

int compararHistogramas(unsigned int* histA, unsigned int* histB){
        int valido = 1;
        for(int i = 0; i<IMDEP; i++){
                if(histA[i] != histB[i]){
                        printf("Error en [%d]: %u != %u\n", i, histA[i], histB[i]);
                        valido = 0;
                }
        }
        return valido;
}


__global__ void kernelHistograma(unsigned char *imagen, unsigned long size, unsigned int* histo){
        __shared__ unsigned int temp[IMDEP];
        temp[threadIdx.x] = 0;
        __syncthreads();

        unsigned long i = threadIdx.x + blockIdx.x * blockDim.x;
        int offset = blockDim.x * gridDim.x;

        while (i < size) {
                atomicAdd( &temp[imagen[i]], 1);
                i += offset;
        }

        __syncthreads();
        atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}

int main(void){
        unsigned char* imagen = (unsigned char*) inicializarImagen(SIZE);
        unsigned int histoCPU[IMDEP];
        histogramaCPU(imagen, SIZE, histoCPU);
        long chk = calcularCheckSum(histoCPU);
        printf("Check-sum CPU: %ld\n", chk);


        unsigned char *dev_imagen = 0;
        unsigned int *dev_histo = 0;
        hipMalloc( (void**) &dev_imagen, SIZE );
        hipMemcpy( dev_imagen, imagen, SIZE, hipMemcpyHostToDevice );
        hipMalloc( (void**) &dev_histo, IMDEP * sizeof( unsigned int) );

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float milliSeconds = 0.0;
        float aveGPUMS = 0.0;

        for(int iter = -1; iter<numRuns; iter++){//La iteraciÃ³n -1 es para que la tarjeta se ponga en marcha, normalmente siempre da peores tiempos.
                hipMemset( dev_histo, 0, IMDEP * sizeof( unsigned int ) );
                if(iter<0){
                        kernelHistograma<<<NBLOCKS, THREADS_PER_BLOCK>>>(dev_imagen, SIZE, dev_histo);
                }else{
                        hipDeviceSynchronize();
                        hipEventRecord(start);
                        kernelHistograma<<<NBLOCKS, THREADS_PER_BLOCK>>>(dev_imagen, SIZE, dev_histo);
                        hipEventRecord(stop);
                        hipEventSynchronize(stop);
                        hipEventElapsedTime(&milliSeconds, start, stop);
                        aveGPUMS += milliSeconds;
                }
        }
        hipEventDestroy(start);
        hipEventDestroy(stop);

        unsigned int gpuHisto[IMDEP];
        hipMemcpy(gpuHisto, dev_histo, IMDEP * sizeof(unsigned int), hipMemcpyDeviceToHost );
        chk = calcularCheckSum(gpuHisto);
        printf("Check-sum GPU: %ld\n", chk);

        if(compararHistogramas(histoCPU, gpuHisto))
                printf("CÃ¡lculo correcto!!\n");

        printf("Tiempo medio de ejecucion del kernel<<<%d, %d>>> sobre %u bytes [s]: %.4f\n", NBLOCKS, THREADS_PER_BLOCK, SIZE, aveGPUMS / 1000.0);

        free(imagen);
        hipFree(dev_imagen);
        hipFree(dev_histo);
        return 0;
}
