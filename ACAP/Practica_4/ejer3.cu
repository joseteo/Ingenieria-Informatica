#include <cstdio>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"

#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <vector>

__global__ void producto_matrices(double* m_a, double* m_b, double* m_r, int n) {
  int fila = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;

  double temp = 0;

  if(fila < n && col < n){
    for(int i=0; i < n; i++){
      temp += m_a[fila * n + i] * m_b[i * n + col];
    }
  }

  m_r[fila * n + col] = temp;
}

int main(void)
{
  printf("Introduce las dimensiones de las matrices (recuerda si las dimensiones son matriz1 AxB y matriz2 CxD, la B y la C han de ser iguales)\n");
  int A=0, B=0, C=0, D=0;

  printf("\nA=");
  A = scanf("%d", &A);
  printf("\nB=");
  B = scanf("%d", &B);
  printf("\nC=");
  C = scanf("%d", &C);
  printf("\nD=");
  D = scanf("%d", &D);

  if(B!=C){
    printf("Introduce las dimensiones de las matrices (recuerda si las dimensiones son matriz1 AxB y matriz2 CxD, la B y la C han de ser iguales)\n");
    printf("\nA=");
    A = scanf("%d", &A);
    printf("\nB=");
    B = scanf("%d", &B);
    printf("\nC=");
    C = scanf("%d", &C);
    printf("\nD=");
    D = scanf("%d", &D);
  }
  int N = (A > B) ? ((A > D) ? A : D) : (B > D) ? B : D; //si el maximo es A si el maximo es B o es D
  int size = N*N;
  double *m_a, *d_m_a, *m_b, *d_m_b, *m_r, *d_m_r;

  //alojar memoria
  m_r = (double*)malloc(sizeof(double));
  m_a = (double*)malloc(A*B * sizeof(double));
  m_b = (double*)malloc(C*D * sizeof(double));
  hipMalloc((void**)&d_m_r, sizeof(double));
  hipMalloc((void**)&d_m_a, A*B * sizeof(double));
  hipMalloc((void**)&d_m_b, C*D * sizeof(double));
  hipMemset(d_m_r, 0, sizeof(double));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for(int i=0; i < A*B; i++)
    m_a[i] = double(rand()%(A*B)) / (A*B);

  for(int i=0; i < C*D; i++)
    m_b[i] = double(rand()%(C*D)) / (C*D);


  hipEventRecord(start);

  hipMemcpy(d_m_a, m_a, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_m_b, m_b, N*sizeof(double), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(N, N);
  dim3 blocksPerGrid(1, 1);
    if (N*N > 512){
      threadsPerBlock.x = 512;
      threadsPerBlock.y = 512;
      blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
      blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
    }

  producto_matrices <<< blocksPerGrid, threadsPerBlock >>> (d_m_a, d_m_b, d_m_r, N); //512 es gridSize, y 512 es blockSize

  hipMemcpy(m_r, d_m_r, size*sizeof(double), hipMemcpyDeviceToHost);

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  for (int i=0; i < A*D; i++)
      printf("%lf\n", *m_r);

  printf("Tiempo GPU: %fms\n", milliseconds);

  hipEventRecord(start);

  double *cpu_m_r;
  cpu_m_r = (double*)malloc(sizeof(double));
  double sum;
  for (int i=0; i<A*D; i++){
    sum += m_a[i]*m_b[i];
    cpu_m_r[i] = sum;
  }

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  for (int i=0; i < size; i++)
    printf("%lf\n", *cpu_m_r);

  printf("Tiempo CPU: %fms\n", milliseconds);

  //liberar memoria
  free(m_r);
  free(d_m_r);
  free(cpu_m_r);
  hipFree(m_a);
  hipFree(d_m_a);
  hipFree(m_b);
  hipFree(d_m_b);
}
