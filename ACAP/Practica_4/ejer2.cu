#include <cstdio>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"

#include <hiprand/hiprand_kernel.h>
#include <ctime>

__global__ void encontrar_maximo(double* numeros, double* max, int* mutex, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = gridDim.x * blockDim.x;    //estos dos permiten hacer que si el tamaño de bloque es 100 y el de hilos es 10,
  int offset = 0;   // con estos el primer thread se encargará de la primera entrada del vector global de la 101 de la 201 en adelante,
                    //lo que lo hace mas flexible y escalable

  __shared__ double cache[512];

  double temp = -1;
  while(i + offset < n){
    temp = fmax(temp, numeros[i + offset]);
    offset += stride;
  }

  cache[threadIdx.x] = temp;

  __syncthreads();

  //reduccion del vector, y al finalizar el mayor valor de entre los 512 se encontrara en la primera posicion de cache
  int j = blockDim.x/2;
  while(j != 0){
    if(threadIdx.x < j){
      cache[threadIdx.x] = fmax(cache[threadIdx.x], cache[threadIdx.x + j]);
    }

    __syncthreads();
    j /= 2;
  }

  if(threadIdx.x == 0){
    while(atomicCAS(mutex, 0, 1) != 0)  // bloquea el mutex //sentencia atomica entre distintas hebras, el mutex se usa para la exclusion,
      *max = fmax(*max, cache[0]);      //y sirve como llave para usar la variable global sin riesgo

      //printf("MAXIMO= %lf\n", *max);
    atomicExch(mutex, 0); //libera el mutex
  }
}
/*
__global__ void cudaRand(double* numeros){
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  curandState state;
  curand_init((unsigned long long)clock(), 0, 0, &state);

  printf("numero[%d] = %lf\n", i, numeros[i]);
  numeros[i] = curand_uniform_double(&state);
}*/

int main(void)
{
  int N = 1024*1024;
  double *x, *d_x, *x_max, *d_x_max;
  int* mutex;

  //alojar memoria
  x = (double*)malloc(N*sizeof(double));
  x_max = (double*)malloc(sizeof(double));
  hipMalloc((void**)&d_x, N * sizeof(double));
  hipMalloc((void**)&d_x_max, sizeof(double));
  hipMalloc((void**)&mutex, sizeof(int));
  hipMemset(mutex, 0, sizeof(double));
  hipMemset(d_x_max, 0, sizeof(double)); //inicializar el maximo a 0 para evitar basura inicial

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  //cudaRand <<< 1, N >>> (d_x); //genera numeros random
  for(int i=0; i<N; i++)
    x[i] = N*double(rand()) / RAND_MAX;

  hipEventRecord(start);

  hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);

  for(int i=0; i<1000; i++)
  encontrar_maximo <<< 512, 512 >>> (d_x, d_x_max, mutex, N); //512 es gridSize, y 512 es blockSize

  hipMemcpy(x_max, d_x_max, sizeof(double), hipMemcpyDeviceToHost);

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);


  /*for (int i = 0; i < N; i++) {
    printf("Elementos Aleatorios Del Vector: %lf\n", x[i]);
  }*/

  printf("Maximo elemento GPU: %lf\n", *x_max);
  printf("Tiempo GPU: %fms\n", milliseconds);

  hipEventRecord(start);

  for(int i=0; i<1000; i++){
    *x_max = -1.0;
    for(int i=0; i<N; i++){
      if(x[i] > *x_max){
        *x_max = x[i];
      }
    }
  }

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("Maximo elemento CPU: %lf\n", *x_max);
  printf("Tiempo CPU: %fms\n", milliseconds);

  //liberar memoria
  free(x);
  free(x_max);
  hipFree(d_x);
  hipFree(d_x_max);
  hipFree(mutex);
}
